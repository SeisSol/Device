#include "hip/hip_runtime.h"
#include "AbstractAPI.h"
#include "interfaces/cuda/Internals.h"
#include <device.h>
#include <cassert>

namespace device {
  __global__ void kernel_streamBatchedData(real **baseSrcPtr,
                                           real **baseDstPtr,
                                           unsigned elementSize) {

    real *srcElement = baseSrcPtr[blockIdx.x];
    real *dstElement = baseDstPtr[blockIdx.x];
    for (int index = threadIdx.x; index < elementSize; index += blockDim.x) {
      dstElement[index] = srcElement[index];
    }
  }

  void Algorithms::streamBatchedData(real **baseSrcPtr,
                                     real **baseDstPtr,
                                     unsigned elementSize,
                                     unsigned numElements) {
    dim3 block(internals::WARP_SIZE, 1, 1);
    dim3 grid(numElements, 1, 1);
    kernel_streamBatchedData<<<grid, block>>>(baseSrcPtr, baseDstPtr, elementSize); CHECK_ERR;
  }


//--------------------------------------------------------------------------------------------------
  __global__ void kernel_accumulateBatchedData(real **baseSrcPtr,
                                               real **baseDstPtr,
                                               unsigned elementSize) {

    real *srcElement = baseSrcPtr[blockIdx.x];
    real *dstElement = baseDstPtr[blockIdx.x];
    for (int index = threadIdx.x; index < elementSize; index += blockDim.x) {
      dstElement[index] += srcElement[index];
    }
  }

  void Algorithms::accumulateBatchedData(real **baseSrcPtr,
                                         real **baseDstPtr,
                                         unsigned elementSize,
                                         unsigned numElements) {
    dim3 block(internals::WARP_SIZE, 1, 1);
    dim3 grid(numElements, 1, 1);
    kernel_accumulateBatchedData<<<grid, block>>>(baseSrcPtr, baseDstPtr, elementSize); CHECK_ERR;
  }

//--------------------------------------------------------------------------------------------------
  __global__ void kernel_touchBatchedMemory(real **basePtr, unsigned elementSize, bool clean) {
    real *element = basePtr[blockIdx.x];
    int id = threadIdx.x;
    while (id < elementSize) {
      if (clean) {
        element[id] = 0.0;
      } else {
        real value = element[id];
        // Do something dummy here. We just need to check the pointers point to valid memory locations.
        // Avoid compiler optimization. Possibly, implement a dummy code with asm.
        value += 1.0;
        value -= 1.0;
      }
      id += blockDim.x;
    }
  }

  void Algorithms::touchBatchedMemory(real **basePtr, unsigned elementSize, unsigned numElements, bool clean) {
    dim3 block(256, 1, 1);
    dim3 grid(numElements, 1, 1);
    kernel_touchBatchedMemory<<<grid, block>>>(basePtr, elementSize, clean); CHECK_ERR;
  }

//--------------------------------------------------------------------------------------------------
  template<typename T>
  __global__ void kernel_copyUniformToScatter(T *src, T **dst, size_t chunkSize) {
    T *srcElement = &src[blockIdx.x];
    T *dstElement = dst[blockIdx.x];
    for (int index = threadIdx.x; index < chunkSize; index += blockDim.x) {
      dstElement[index] = srcElement[index];
    }
  }

  template<typename T>
  void Algorithms::copyUniformToScatter(T *src, T **dst, size_t chunkSize, size_t numElements, void* streamPtr) {
    dim3 block(256, 1, 1);
    dim3 grid(numElements, 1, 1);
    hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
    kernel_copyUniformToScatter<<<grid, block, 0, stream>>>(src, dst, chunkSize); CHECK_ERR;
    CHECK_ERR;
  }
  template void Algorithms::copyUniformToScatter(real *src, real **dst, size_t chunkSize, size_t numElements, void* streamPtr);
  template void Algorithms::copyUniformToScatter(int *src, int **dst, size_t chunkSize, size_t numElements, void* streamPtr);
  template void Algorithms::copyUniformToScatter(char *src, char **dst, size_t chunkSize, size_t numElements, void* streamPtr);

//--------------------------------------------------------------------------------------------------
  template<typename T>
  __global__ void kernel_copyScatterToUniform(T **src, T *dst, size_t chunkSize) {
    T *srcElement = src[blockIdx.x];
    T *dstElement = &dst[blockIdx.x];
    for (int index = threadIdx.x; index < chunkSize; index += blockDim.x) {
      dstElement[index] = srcElement[index];
    }
  }

  template<typename T>
  void Algorithms::copyScatterToUniform(T **src, T *dst, size_t chunkSize, size_t numElements, void* streamPtr) {
    dim3 block(256, 1, 1);
    dim3 grid(numElements, 1, 1);
    hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
    kernel_copyScatterToUniform<<<grid, block, 0, stream>>>(src, dst, chunkSize); CHECK_ERR;
    CHECK_ERR;
  }
  template void Algorithms::copyScatterToUniform(real **src, real *dst, size_t chunkSize, size_t numElements, void* streamPtr);
  template void Algorithms::copyScatterToUniform(int **src, int *dst, size_t chunkSize, size_t numElements, void* streamPtr);
  template void Algorithms::copyScatterToUniform(char **src, char *dst, size_t chunkSize, size_t numElements, void* streamPtr);
} // namespace device
