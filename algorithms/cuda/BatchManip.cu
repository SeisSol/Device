#include "hip/hip_runtime.h"
#include "AbstractAPI.h"
#include "interfaces/cuda/Internals.h"
#include <device.h>
#include <cassert>

namespace device {
  __global__ void kernel_streamBatchedData(real **baseSrcPtr,
                                           real **baseDstPtr,
                                           unsigned elementSize) {

    real *srcElement = baseSrcPtr[blockIdx.x];
    real *dstElement = baseDstPtr[blockIdx.x];
    for (int index = threadIdx.x; index < elementSize; index += blockDim.x) {
      dstElement[index] = srcElement[index];
    }
  }

  void Algorithms::streamBatchedData(real **baseSrcPtr,
                                     real **baseDstPtr,
                                     unsigned elementSize,
                                     unsigned numElements,
                                     void* streamPtr) {
    dim3 block(internals::WARP_SIZE, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_streamBatchedData<<<grid, block, 0, stream>>>(baseSrcPtr, baseDstPtr, elementSize); CHECK_ERR;
  }


//--------------------------------------------------------------------------------------------------
  __global__ void kernel_accumulateBatchedData(real **baseSrcPtr,
                                               real **baseDstPtr,
                                               unsigned elementSize) {

    real *srcElement = baseSrcPtr[blockIdx.x];
    real *dstElement = baseDstPtr[blockIdx.x];
    for (int index = threadIdx.x; index < elementSize; index += blockDim.x) {
      dstElement[index] += srcElement[index];
    }
  }

  void Algorithms::accumulateBatchedData(real **baseSrcPtr,
                                         real **baseDstPtr,
                                         unsigned elementSize,
                                         unsigned numElements,
                                         void* streamPtr) {
    dim3 block(internals::WARP_SIZE, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_accumulateBatchedData<<<grid, block, 0, stream>>>(baseSrcPtr, baseDstPtr, elementSize); CHECK_ERR;
  }

//--------------------------------------------------------------------------------------------------
  __global__ void kernel_touchBatchedMemory(real **basePtr, unsigned elementSize, bool clean) {
    real *element = basePtr[blockIdx.x];
    int id = threadIdx.x;
    while (id < elementSize) {
      if (clean) {
        element[id] = 0.0;
      } else {
        real value = element[id];
        // Do something dummy here. We just need to check the pointers point to valid memory locations.
        // Avoid compiler optimization. Possibly, implement a dummy code with asm.
        value += 1.0;
        value -= 1.0;
      }
      id += blockDim.x;
    }
  }

  void Algorithms::touchBatchedMemory(real **basePtr,
                                      unsigned elementSize,
                                      unsigned numElements,
                                      bool clean,
                                      void* streamPtr) {
    dim3 block(256, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_touchBatchedMemory<<<grid, block, 0, stream>>>(basePtr, elementSize, clean); CHECK_ERR;
  }

//--------------------------------------------------------------------------------------------------
  template<typename T>
  __global__ void kernel_copyUniformToScatter(T *src, T **dst, size_t srcOffset, size_t copySize) {
    T *srcElement = &src[blockIdx.x * srcOffset];
    T *dstElement = dst[blockIdx.x];
    for (int index = threadIdx.x; index < copySize; index += blockDim.x) {
      dstElement[index] = srcElement[index];
    }
  }

  template<typename T>
  void Algorithms::copyUniformToScatter(T *src,
                                        T **dst,
                                        size_t srcOffset,
                                        size_t copySize,
                                        size_t numElements,
                                        void* streamPtr) {
    dim3 block(256, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_copyUniformToScatter<<<grid, block, 0, stream>>>(src, dst, srcOffset, copySize); CHECK_ERR;
    CHECK_ERR;
  }
  template void Algorithms::copyUniformToScatter(real *src,
                                                 real **dst,
                                                 size_t srcOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

  template void Algorithms::copyUniformToScatter(int *src,
                                                 int **dst,
                                                 size_t srcOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

  template void Algorithms::copyUniformToScatter(char *src,
                                                 char **dst,
                                                 size_t srcOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

//--------------------------------------------------------------------------------------------------
  template<typename T>
  __global__ void kernel_copyScatterToUniform(T **src, T *dst, size_t dstOffset, size_t copySize) {
    T *srcElement = src[blockIdx.x];
    T *dstElement = &dst[blockIdx.x * dstOffset];
    for (int index = threadIdx.x; index < copySize; index += blockDim.x) {
      dstElement[index] = srcElement[index];
    }
  }

  template<typename T>
  void Algorithms::copyScatterToUniform(T **src,
                                        T *dst,
                                        size_t dstOffset,
                                        size_t copySize,
                                        size_t numElements,
                                        void* streamPtr) {
    dim3 block(256, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_copyScatterToUniform<<<grid, block, 0, stream>>>(src, dst, dstOffset, copySize); CHECK_ERR;
    CHECK_ERR;
  }
  template void Algorithms::copyScatterToUniform(real **src,
                                                 real *dst,
                                                 size_t dstOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

  template void Algorithms::copyScatterToUniform(int **src,
                                                 int *dst,
                                                 size_t dstOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

  template void Algorithms::copyScatterToUniform(char **src,
                                                 char *dst,
                                                 size_t dstOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);
} // namespace device
