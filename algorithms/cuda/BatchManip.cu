#include "hip/hip_runtime.h"
#include "AbstractAPI.h"
#include "interfaces/cuda/Internals.h"
#include <device.h>
#include <cassert>

namespace device {
  __global__ void kernel_streamBatchedData(real **baseSrcPtr,
                                           real **baseDstPtr,
                                           unsigned elementSize) {

    real *srcElement = baseSrcPtr[blockIdx.x];
    real *dstElement = baseDstPtr[blockIdx.x];
    for (int index = threadIdx.x; index < elementSize; index += blockDim.x) {
      dstElement[index] = srcElement[index];
    }
  }

  void Algorithms::streamBatchedData(real **baseSrcPtr,
                                     real **baseDstPtr,
                                     unsigned elementSize,
                                     unsigned numElements,
                                     void* streamPtr) {
    dim3 block(internals::WARP_SIZE, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_streamBatchedData<<<grid, block, 0, stream>>>(baseSrcPtr, baseDstPtr, elementSize); CHECK_ERR;
  }


//--------------------------------------------------------------------------------------------------
  __global__ void kernel_accumulateBatchedData(real **baseSrcPtr,
                                               real **baseDstPtr,
                                               unsigned elementSize) {

    real *srcElement = baseSrcPtr[blockIdx.x];
    real *dstElement = baseDstPtr[blockIdx.x];
    for (int index = threadIdx.x; index < elementSize; index += blockDim.x) {
      dstElement[index] += srcElement[index];
    }
  }

  void Algorithms::accumulateBatchedData(real **baseSrcPtr,
                                         real **baseDstPtr,
                                         unsigned elementSize,
                                         unsigned numElements,
                                         void* streamPtr) {
    dim3 block(internals::WARP_SIZE, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_accumulateBatchedData<<<grid, block, 0, stream>>>(baseSrcPtr, baseDstPtr, elementSize); CHECK_ERR;
  }

//--------------------------------------------------------------------------------------------------
  __global__ void kernel_touchBatchedMemory(real **basePtr, unsigned elementSize, bool clean) {
    real *element = basePtr[blockIdx.x];
    int id = threadIdx.x;
    while (id < elementSize) {
      if (clean) {
        element[id] = 0.0;
      } else {
        real value = element[id];
        // Do something dummy here. We just need to check the pointers point to valid memory locations.
        // Avoid compiler optimization. Possibly, implement a dummy code with asm.
        value += 1.0;
        value -= 1.0;
      }
      id += blockDim.x;
    }
  }

  void Algorithms::touchBatchedMemory(real **basePtr,
                                      unsigned elementSize,
                                      unsigned numElements,
                                      bool clean,
                                      void* streamPtr) {
    dim3 block(256, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_touchBatchedMemory<<<grid, block, 0, stream>>>(basePtr, elementSize, clean); CHECK_ERR;
  }

//--------------------------------------------------------------------------------------------------
__global__  void kernel_setToValue(real** out, real value, size_t elementSize, size_t numElements) {
  const int elementId = blockIdx.x;
  if (elementId < numElements) {
    real *element = out[elementId];
    const int tid = threadIdx.x;
    for (int i = tid; i < elementSize; i += blockDim.x) {
      element[i] = value;
    }
  }
}

void Algorithms::setToValue(real** out, real value, size_t elementSize, size_t numElements, void* streamPtr) {
  dim3 block(256, 1, 1);
  dim3 grid(numElements, 1, 1);
  auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
  kernel_setToValue<<<grid, block, 0, stream>>>(out, value, elementSize, numElements);
  CHECK_ERR;
}

//--------------------------------------------------------------------------------------------------
  template<typename T>
  __global__ void kernel_copyUniformToScatter(T *src, T **dst, size_t srcOffset, size_t copySize) {
    T *srcElement = &src[blockIdx.x * srcOffset];
    T *dstElement = dst[blockIdx.x];
    for (int index = threadIdx.x; index < copySize; index += blockDim.x) {
      dstElement[index] = srcElement[index];
    }
  }

  template<typename T>
  void Algorithms::copyUniformToScatter(T *src,
                                        T **dst,
                                        size_t srcOffset,
                                        size_t copySize,
                                        size_t numElements,
                                        void* streamPtr) {
    dim3 block(256, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_copyUniformToScatter<<<grid, block, 0, stream>>>(src, dst, srcOffset, copySize); CHECK_ERR;
    CHECK_ERR;
  }
  template void Algorithms::copyUniformToScatter(real *src,
                                                 real **dst,
                                                 size_t srcOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

  template void Algorithms::copyUniformToScatter(int *src,
                                                 int **dst,
                                                 size_t srcOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

  template void Algorithms::copyUniformToScatter(char *src,
                                                 char **dst,
                                                 size_t srcOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

//--------------------------------------------------------------------------------------------------
  template<typename T>
  __global__ void kernel_copyScatterToUniform(T **src, T *dst, size_t dstOffset, size_t copySize) {
    T *srcElement = src[blockIdx.x];
    T *dstElement = &dst[blockIdx.x * dstOffset];
    for (int index = threadIdx.x; index < copySize; index += blockDim.x) {
      dstElement[index] = srcElement[index];
    }
  }

  template<typename T>
  void Algorithms::copyScatterToUniform(T **src,
                                        T *dst,
                                        size_t dstOffset,
                                        size_t copySize,
                                        size_t numElements,
                                        void* streamPtr) {
    dim3 block(256, 1, 1);
    dim3 grid(numElements, 1, 1);
    auto stream = reinterpret_cast<internals::deviceStreamT>(streamPtr);
    kernel_copyScatterToUniform<<<grid, block, 0, stream>>>(src, dst, dstOffset, copySize); CHECK_ERR;
    CHECK_ERR;
  }
  template void Algorithms::copyScatterToUniform(real **src,
                                                 real *dst,
                                                 size_t dstOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

  template void Algorithms::copyScatterToUniform(int **src,
                                                 int *dst,
                                                 size_t dstOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);

  template void Algorithms::copyScatterToUniform(char **src,
                                                 char *dst,
                                                 size_t dstOffset,
                                                 size_t copySize,
                                                 size_t numElements,
                                                 void* streamPtr);
} // namespace device
