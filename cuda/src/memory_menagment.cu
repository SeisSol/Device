#include "device_utils.h"



void * device_malloc(size_t size) {
    void *devPtr;
    hipMalloc(&devPtr, size); CUDA_CHECK;
    return devPtr;
}


void device_copy_to(void* dst, const void* src, size_t count) {
    hipMemcpy(dst, src, count, hipMemcpyHostToDevice); CUDA_CHECK;
}


void device_copy_from(void* dst, const void* src, size_t count) {
    hipMemcpy(dst, src, count, hipMemcpyDeviceToHost); CUDA_CHECK;
}


void device_free(void *devPtr) {
    hipFree(devPtr); CUDA_CHECK;
}