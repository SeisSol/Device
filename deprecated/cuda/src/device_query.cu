#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace device {
  namespace query {

    void init() {
      hipInit(0); CUDA_CHECK;
    }

    int getNumDevices() {
      int numDevices{};
      hipGetDeviceCount(&numDevices); CUDA_CHECK;
      return numDevices;
    }

    void setDevice(int device_id) {
      hipSetDevice(device_id); CUDA_CHECK;
    }
  }
}