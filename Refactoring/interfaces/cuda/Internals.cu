#include <iostream>
#include <hip/hip_runtime.h>

namespace device {
  namespace internals {

    std::string PrevFile = "";
    int PrevLine = 0;

    void checkErr(const std::string &file, int line) {
      hipError_t Error = hipGetLastError();
      if (Error != hipSuccess) {
        std::cout << std::endl << file << ", line " << line
                  << ": " << hipGetErrorString(Error) << " (" << Error << ")" << std::endl;
        if (PrevLine > 0)
          std::cout << "Previous CUDA call:" << std::endl
                    << PrevFile << ", line " << PrevLine << std::endl;
        throw;
      }
      PrevFile = file;
      PrevLine = line;
    }
  }
}

