#include "hip/hip_runtime.h"
#include "assert.h"

#include "CudaInterface.h"
#include "Internals.h"
#include <iostream>

using namespace device;

unsigned ConcreteInterface::createStream(StreamType Type) {
  static size_t StreamIdCounter = 1;

  hipStream_t* Stream = new hipStream_t;
  hipStreamCreateWithFlags(Stream,
                            Type == StreamType::Blocking ? hipStreamDefault : hipStreamNonBlocking); CHECK_ERR;
  unsigned StreamId = StreamIdCounter;

  assert((m_IdToStreamMap.find(StreamId) == m_IdToStreamMap.end())
          && (m_IdToStreamMap.find(0) == m_IdToStreamMap.end())
          && "DEVICE:: overflow w.r.t. the number of streams");

  m_IdToStreamMap[StreamId] = Stream;

  ++StreamIdCounter;
  return StreamId;
}


void ConcreteInterface::deleteStream(unsigned StreamId) {
  assert((m_IdToStreamMap.find(StreamId) != m_IdToStreamMap.end()) && "DEVICE: a stream doesn't exist");
  hipStreamDestroy(*m_IdToStreamMap[StreamId]);
  m_IdToStreamMap.erase(StreamId);
}


void ConcreteInterface::deleteAllCreatedStreams() {
  for (auto& Stream: m_IdToStreamMap) {
    hipStreamDestroy(*(Stream.second)); CHECK_ERR;
  }
  m_IdToStreamMap.erase(m_IdToStreamMap.begin(), m_IdToStreamMap.end());
  m_CurrentComputeStream = m_DefaultStream;
}


void ConcreteInterface::setComputeStream(unsigned StreamId) {
  assert((m_IdToStreamMap.find(StreamId) != m_IdToStreamMap.end()) && "DEVICE: a stream doesn't exist");
  m_CurrentComputeStream = *m_IdToStreamMap[StreamId];
}


void ConcreteInterface::setDefaultComputeStream() {
  m_CurrentComputeStream = m_DefaultStream;
}

__global__ void kernel_synchAllStreams() {
  // NOTE: an empty stream. It is supposed to get called with Cuda default stream. It is going to force all
  // other streams to finish their tasks
}


void ConcreteInterface::synchAllStreams() {
  kernel_synchAllStreams<<<1,1>>>();
}