#include "CudaInterface.h"
#include "Internals.h"

using namespace device;


void* ConcreteInterface::allocGlobMem(size_t Size) {
  void *devPtr;
  hipMalloc(&devPtr, Size); CHECK_ERR;
  return devPtr;
}


void* ConcreteInterface::allocUnifiedMem(size_t Size) {
  void *devPtr;
  hipMallocManaged(&devPtr, Size, hipMemAttachGlobal); CHECK_ERR;
  return devPtr;
}


void* ConcreteInterface::allocPinnedMem(size_t Size) {
  void *devPtr;
  hipHostMalloc(&devPtr, Size); CHECK_ERR;
  return devPtr;
}


void ConcreteInterface::freeMem(void *DevPtr) {
  hipFree(DevPtr); CHECK_ERR;
}


void ConcreteInterface::freePinnedMem(void *DevPtr) {
  hipHostFree(DevPtr); CHECK_ERR;
}



char* ConcreteInterface::getTempMemory() {

}


void ConcreteInterface::freeTempMemory() {
  
}
