#include "hip/hip_runtime.h"
#include "CudaInterface.h"
#include "Internals.h"

using namespace device;


void ConcreteInterface::copyTo(void* Dst, const void* Src, size_t Count) {
  hipMemcpy(Dst, Src, Count, hipMemcpyHostToDevice); CHECK_ERR;
}


void ConcreteInterface::copyFrom(void* Dst, const void* Src, size_t Count) {
  hipMemcpy(Dst, Src, Count, hipMemcpyDeviceToHost); CHECK_ERR;
}


void ConcreteInterface::copyBetween(void* Dst, const void* Src, size_t Count) {
  hipMemcpy(Dst, Src, Count, hipMemcpyDeviceToDevice); CHECK_ERR;
}


void ConcreteInterface::copy2dArrayTo(void *Dst,
                                      size_t Dpitch,
                                      const void *Src,
                                      size_t Spitch,
                                      size_t Width,
                                      size_t Height) {
  hipMemcpy2D(Dst, Dpitch, Src, Spitch, Width, Height, hipMemcpyHostToDevice); CHECK_ERR;
}


void ConcreteInterface::copy2dArrayFrom(void *Dst,
                                        size_t Dpitch,
                                        const void *Src,
                                        size_t Spitch,
                                        size_t Width,
                                        size_t Height) {
  hipMemcpy2D(Dst, Dpitch, Src, Spitch, Width, Height, hipMemcpyDeviceToHost); CHECK_ERR;
}


__global__ void kernel_streamBatchedData(real **BaseSrcPtr,
                                         real **BaseDstPtr,
                                         unsigned ElementSize,
                                         bool Accumulate) {

  real *SrcElement = BaseSrcPtr[blockIdx.x];
  real *DstElement = BaseDstPtr[blockIdx.x];

  int Id = threadIdx.x;
  while (Id < ElementSize) {
    if (Accumulate) {
      DstElement[Id] += SrcElement[Id];
    } else {
      DstElement[Id] = SrcElement[Id];
    }
    Id += blockDim.x;
  }
}

void ConcreteInterface::streamBatchedData(real **BaseSrcPtr,
                                          real **BaseDstPtr,
                                          unsigned ElementSize,
                                          unsigned NumElements) {
  dim3 Block(256, 1, 1);
  dim3 Grid(NumElements, 1, 1);
  kernel_streamBatchedData<<<Grid, Block>>>(BaseSrcPtr, BaseDstPtr, ElementSize, false); CHECK_ERR;
}


void ConcreteInterface::accumulateBatchedData(real **BaseSrcPtr,
                                              real **BaseDstPtr,
                                              unsigned ElementSize,
                                              unsigned NumElements) {
  dim3 Block(ElementSize, 1, 1);
  dim3 Grid(NumElements, 1, 1);
  kernel_streamBatchedData<<<Grid, Block>>>(BaseSrcPtr, BaseDstPtr, ElementSize, true); CHECK_ERR;
}