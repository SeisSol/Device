#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2020-2024 SeisSol Group
//
// SPDX-License-Identifier: BSD-3-Clause

#include "CudaWrappedAPI.h"
#include "Internals.h"

namespace device {
__global__ void kernel_checkOffloading() { printf("gpu offloading is working\n"); }

void ConcreteAPI::checkOffloading() {
  isFlagSet<StatusID::DeviceSelected>(status);
  kernel_checkOffloading<<<1, 1>>>();
  CHECK_ERR;
  hipDeviceSynchronize();
}
} // namespace device

