#include "hip/hip_runtime.h"
#include "CudaWrappedAPI.h"
#include "Internals.h"

namespace device {
__global__ void kernel_checkOffloading() { printf("gpu offloading is working\n"); }

void ConcreteAPI::checkOffloading() {
  isFlagSet<StatusId::DeviceSelected>();
  kernel_checkOffloading<<<1, 1>>>();
  CHECK_ERR;
  hipDeviceSynchronize();
}
} // namespace device