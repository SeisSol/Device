#include <iostream>
#include <string>
#include <sstream>
#include <hip/hip_runtime.h>

#ifdef PROFILING_ENABLED
#include <nvToolsExt.h>
#endif

#include "CudaWrappedAPI.h"
#include "Internals.h"

using namespace device;

void ConcreteAPI::initialize() {
  hipInit(0); CHECK_ERR;
}

void ConcreteAPI::allocateStackMem() {

  // try to detect the amount of temp. memory from the environment
  const size_t Factor = 1024 * 1024 * 1024;  //!< bytes in 1 GB
  std::ostringstream Info;
  try {
    char *ValueString = std::getenv("DEVICE_STACK_MEM_SIZE");
    if (!ValueString) {
      Info << "DEVICE::INFO env. variable \"DEVICE_STACK_MEM_SIZE\" has not been set. "
           << "The default amount of the device memory (1 GB) "
           << "is going to be used to store temp. variables during execution of compute-kernels\n";
    }
    else {
      double RequestedStackMem = std::stod(std::string(ValueString));
      m_MaxStackMem = Factor * RequestedStackMem;
      Info << "DEVICE::INFO: env. variable \"DEVICE_STACK_MEM_SIZE\" has been detected. "
           << RequestedStackMem << "GB of the device memory is going to be used "
           << "to store temp. variables during execution of compute-kernels\n";
    }
  }
  catch (const std::invalid_argument &Err) {
    std::cout << "DEVICE::ERROR: " << Err.what() << ". File: " << __FILE__ << ", line: " << __LINE__ << '\n';
    throw Err;
  }
  catch (const std::out_of_range& Err) {
    std::cout << "DEVICE::ERROR: " << Err.what() << ". File: " << __FILE__ << ", line: " << __LINE__ << '\n';
    throw Err;
  }

  //TODO: move info into SeisSol logger
  std::cout << Info.str() << std::endl;
  hipMalloc(&m_StackMemory, m_MaxStackMem); CHECK_ERR;
};


void ConcreteAPI::finalize() {
  hipFree(m_StackMemory); CHECK_ERR;
  m_StackMemory = nullptr;
};


void ConcreteAPI::setDevice(int DeviceId) {
  std::cout << DeviceId << std::endl;
  m_CurrentDeviceId = DeviceId;
  hipSetDevice(m_CurrentDeviceId); CHECK_ERR;
}


int ConcreteAPI::getNumDevices() {
  int numDevices{};
  hipGetDeviceCount(&numDevices); CHECK_ERR;
  return numDevices;
}


unsigned ConcreteAPI::getMaxThreadBlockSize() {
  int BlockSize{};
  hipDeviceGetAttribute(&BlockSize, hipDeviceAttributeMaxThreadsPerBlock, m_CurrentDeviceId); CHECK_ERR;
  return static_cast<unsigned>(BlockSize);
}


unsigned ConcreteAPI::getMaxSharedMemSize() {
  int SharedMemSize{};
  hipDeviceGetAttribute(&SharedMemSize, hipDeviceAttributeMaxSharedMemoryPerBlock, m_CurrentDeviceId); CHECK_ERR;
  return static_cast<unsigned>(SharedMemSize);
}


unsigned ConcreteAPI::getGlobMemAlignment() {
  // TODO: use hipDeviceGetAttribute
  return 256;
}


void ConcreteAPI::synchDevice() {
  hipDeviceSynchronize(); CHECK_ERR;
}


std::string ConcreteAPI::getDeviceInfoAsText(int DeviceId) {
  hipDeviceProp_t Property;
  hipGetDeviceProperties(&Property, DeviceId); CHECK_ERR;

  std::ostringstream Info;
  Info << "Name: " << Property.name << '\n';
  Info << "totalGlobalMem: " << Property.totalGlobalMem << '\n';
  Info << "sharedMemPerBlock: " << Property.sharedMemPerBlock << '\n';
  Info << "regsPerBlock: " << Property.regsPerBlock << '\n';
  Info << "warpSize: " << Property.warpSize << '\n';
  Info << "memPitch: " << Property.memPitch << '\n';
  Info << "maxThreadsPerBlock: " << Property.maxThreadsPerBlock << '\n';
  Info << "totalConstMem: " << Property.totalConstMem << '\n';
  Info << "clockRate: " << Property.clockRate << '\n';
  Info << "multiProcessorCount: " << Property.multiProcessorCount << '\n';
  Info << "integrated: " << Property.integrated << '\n';
  Info << "canMapHostMemory: " << Property.canMapHostMemory << '\n';
  Info << "computeMode: " << Property.computeMode << '\n';
  Info << "concurrentKernels: " << Property.concurrentKernels << '\n';
  Info << "pciBusID: " << Property.pciBusID << '\n';
  Info << "pciDeviceID: " << Property.pciDeviceID << '\n';

  return Info.str();
}

void ConcreteAPI::putProfilingMark(const std::string &Name, ProfilingColors Color) {
#ifdef PROFILING_ENABLED
  nvtxEventAttributes_t eventAttrib = {0};
  eventAttrib.version = NVTX_VERSION;
  eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  eventAttrib.colorType = NVTX_COLOR_ARGB;
  eventAttrib.color = static_cast<uint32_t>(Color);
  eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
  eventAttrib.message.ascii = Name.c_str();
  nvtxRangePushEx(&eventAttrib);
#endif
}


void ConcreteAPI::popLastProfilingMark() {
#ifdef PROFILING_ENABLED
  nvtxRangePop();
#endif
}