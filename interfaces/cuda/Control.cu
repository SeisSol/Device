// SPDX-FileCopyrightText: 2020-2024 SeisSol Group
//
// SPDX-License-Identifier: BSD-3-Clause

#include "utils/logger.h"
#include "utils/env.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <string>

#ifdef PROFILING_ENABLED
#include <nvToolsExt.h>
#endif

#include "CudaWrappedAPI.h"
#include "Internals.h"

using namespace device;

ConcreteAPI::ConcreteAPI() {
  hipInit(0);
  CHECK_ERR;
  status[StatusID::DriverApiInitialized] = true;
}

void ConcreteAPI::setDevice(int deviceId) {
  currentDeviceId = deviceId;
  hipSetDevice(currentDeviceId);
  CHECK_ERR;

  // Note: the following sets the initial CUDA context
  hipFree(nullptr);
  CHECK_ERR;

  int result;
  hipDeviceGetAttribute(&result, hipDeviceAttributeDirectManagedMemAccessFromHost, currentDeviceId);
  usmDefault = result != 0;

  status[StatusID::DeviceSelected] = true;
}

bool ConcreteAPI::isUnifiedMemoryDefault() {
  return usmDefault;
}

void ConcreteAPI::initialize() {
  if (!status[StatusID::DeviceSelected]) {
    logError() << "Device has not been selected. Please, select device before calling initialize";
  }
  if (!status[StatusID::InterfaceInitialized]) {
    status[StatusID::InterfaceInitialized] = true;
    hipStreamCreateWithFlags(&defaultStream, hipStreamNonBlocking); CHECK_ERR;
    hipEventCreate(&defaultStreamEvent); CHECK_ERR;

    int result{0};
    hipDeviceGetAttribute(&result, hipDeviceAttributeConcurrentManagedAccess, currentDeviceId);
    CHECK_ERR;
    allowedConcurrentManagedAccess = result != 0;

    hipDeviceGetStreamPriorityRange(&priorityMin, &priorityMax);
    CHECK_ERR;
  }
  else {
    logWarning() << "Device Interface has already been initialized";
  }
}

void ConcreteAPI::finalize() {
  if (status[StatusID::InterfaceInitialized]) {
    hipStreamDestroy(defaultStream); CHECK_ERR;
    hipEventDestroy(defaultStreamEvent); CHECK_ERR;
    if (!genericStreams.empty()) {
      printer.printInfo() << "DEVICE::WARNING:" << genericStreams.size()
                               << "device generic stream(s) were not deleted.";
      for (auto stream : genericStreams) {
        hipStreamDestroy(stream); CHECK_ERR;
      }
    }
    status[StatusID::InterfaceInitialized] = false;
  }
}

int ConcreteAPI::getNumDevices() {
  int numDevices{};
  hipGetDeviceCount(&numDevices);
  CHECK_ERR;
  return numDevices;
}

int ConcreteAPI::getDeviceId() {
  if (!status[StatusID::DeviceSelected]) {
    logError() << "Device has not been selected. Please, select device before requesting device Id";
  }
  return currentDeviceId;
}

unsigned ConcreteAPI::getGlobMemAlignment() {
  // TODO: use hipDeviceGetAttribute
  return 128;
}

void ConcreteAPI::syncDevice() {
  isFlagSet<DeviceSelected>(status);
  hipDeviceSynchronize();
  CHECK_ERR;
}

std::string ConcreteAPI::getDeviceInfoAsText(int deviceId) {
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, deviceId);
  CHECK_ERR;

  std::ostringstream info;
  info << "Name: " << property.name << '\n';
  info << "totalGlobalMem: " << property.totalGlobalMem << '\n';
  info << "sharedMemPerBlock: " << property.sharedMemPerBlock << '\n';
  info << "regsPerBlock: " << property.regsPerBlock << '\n';
  info << "warpSize: " << property.warpSize << '\n';
  info << "memPitch: " << property.memPitch << '\n';
  info << "maxThreadsPerBlock: " << property.maxThreadsPerBlock << '\n';
  info << "totalConstMem: " << property.totalConstMem << '\n';
  info << "clockRate: " << property.clockRate << '\n';
  info << "multiProcessorCount: " << property.multiProcessorCount << '\n';
  info << "integrated: " << property.integrated << '\n';
  info << "canMapHostMemory: " << property.canMapHostMemory << '\n';
  info << "computeMode: " << property.computeMode << '\n';
  info << "concurrentKernels: " << property.concurrentKernels << '\n';
  info << "pciBusID: " << property.pciBusID << '\n';
  info << "pciDeviceID: " << property.pciDeviceID << '\n';

  return info.str();
}

std::string ConcreteAPI::getApiName() {
  return "CUDA";
}

std::string ConcreteAPI::getDeviceName(int deviceId) {
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, deviceId);
  CHECK_ERR;

  return property.name;
}

std::string ConcreteAPI::getPciAddress(int deviceId) {
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, deviceId);
  CHECK_ERR;

  std::ostringstream str;
  str << std::setfill('0') << std::setw(4) << std::hex << property.pciDomainID << ":" << std::setw(2) << property.pciBusID << ":" << property.pciDeviceID << "." << "0";
  return str.str();
}

void ConcreteAPI::profilingMessage(const std::string& message) {
#ifdef PROFILING_ENABLED
  isFlagSet<DeviceSelected>(status);
  nvtxMark(message.c_str());
#endif
}

void ConcreteAPI::putProfilingMark(const std::string &name, ProfilingColors color) {
#ifdef PROFILING_ENABLED
  isFlagSet<DeviceSelected>(status);
  nvtxEventAttributes_t eventAttrib = {0};
  eventAttrib.version = NVTX_VERSION;
  eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  eventAttrib.colorType = NVTX_COLOR_ARGB;
  eventAttrib.color = static_cast<uint32_t>(color);
  eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
  eventAttrib.message.ascii = name.c_str();
  nvtxRangePushEx(&eventAttrib);
#endif
}

void ConcreteAPI::popLastProfilingMark() {
#ifdef PROFILING_ENABLED
  isFlagSet<DeviceSelected>(status);
  nvtxRangePop();
#endif
}

void ConcreteAPI::setupPrinting(int rank) {
  printer.setRank(rank);
}

