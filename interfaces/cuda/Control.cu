// SPDX-FileCopyrightText: 2020-2024 SeisSol Group
//
// SPDX-License-Identifier: BSD-3-Clause

#include "utils/logger.h"
#include "utils/env.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <string>

#ifdef PROFILING_ENABLED
#include <nvToolsExt.h>
#endif

#include "CudaWrappedAPI.h"
#include "Internals.h"

using namespace device;

ConcreteAPI::ConcreteAPI() {
  hipInit(0);
  CHECK_ERR;
  status[StatusID::DriverApiInitialized] = true;
}

void ConcreteAPI::setDevice(int deviceId) {
  currentDeviceId = deviceId;
  hipSetDevice(currentDeviceId);
  CHECK_ERR;

  // Note: the following sets the initial CUDA context
  hipFree(nullptr);
  CHECK_ERR;

  int result;
  hipDeviceGetAttribute(&result, hipDeviceAttributeDirectManagedMemAccessFromHost, currentDeviceId);
  usmDefault = result != 0;

  status[StatusID::DeviceSelected] = true;
}

bool ConcreteAPI::isUnifiedMemoryDefault() {
  return usmDefault;
}

void ConcreteAPI::initialize() {
  if (!status[StatusID::DeviceSelected]) {
    logError() << "Device has not been selected. Please, select device before calling initialize";
  }
  if (!status[StatusID::InterfaceInitialized]) {
    status[StatusID::InterfaceInitialized] = true;
    hipStreamCreateWithFlags(&defaultStream, hipStreamNonBlocking); CHECK_ERR;
    hipEventCreate(&defaultStreamEvent); CHECK_ERR;

    int result{0};
    hipDeviceGetAttribute(&result, hipDeviceAttributeConcurrentManagedAccess, currentDeviceId);
    CHECK_ERR;
    allowedConcurrentManagedAccess = result != 0;

    hipDeviceGetStreamPriorityRange(&priorityMin, &priorityMax);
    CHECK_ERR;
  }
  else {
    logWarning() << "Device Interface has already been initialized";
  }
}

void ConcreteAPI::allocateStackMem() {
  isFlagSet<StatusID::DeviceSelected>(status);

  // try to detect the amount of temp. memory from the environment
  const size_t factor = 1024 * 1024 * 1024; //!< bytes in 1 GB

  try {
    char *valueString = std::getenv("DEVICE_STACK_MEM_SIZE");
    if (!valueString) {
      printer.printInfo()
          << "From device: env. variable \"DEVICE_STACK_MEM_SIZE\" has not been set. "
          << "The default amount of the device memory (1 GB) "
          << "is going to be used to store temp. variables during execution of compute-algorithms.";
    } else {
      double requestedStackMem = std::stod(std::string(valueString));
      maxStackMem = factor * requestedStackMem;
      printer.printInfo() << "From device: env. variable \"DEVICE_STACK_MEM_SIZE\" has been detected. "
                    << requestedStackMem << "GB of the device memory is going to be used "
                    << "to store temp. variables during execution of compute-algorithms.";
    }
  } catch (const std::invalid_argument &err) {
    logError() << "DEVICE::ERROR: " << err.what() << ". File: " << __FILE__
               << ", line: " << __LINE__;
  } catch (const std::out_of_range &err) {
    logError() << "DEVICE::ERROR: " << err.what() << ". File: " << __FILE__
               << ", line: " << __LINE__;
  }

  hipMalloc(&stackMemory, maxStackMem);
  CHECK_ERR;

  status[StatusID::StackMemAllocated] = true;
}

void ConcreteAPI::finalize() {
  if (status[StatusID::StackMemAllocated]) {
    hipFree(stackMemory);
    CHECK_ERR;
    stackMemory = nullptr;
    stackMemByteCounter = 0;
    stackMemMeter = std::stack<size_t>{};
    status[StatusID::StackMemAllocated] = false;

  }

  if (status[StatusID::InterfaceInitialized]) {
    hipStreamDestroy(defaultStream); CHECK_ERR;
    hipEventDestroy(defaultStreamEvent); CHECK_ERR;
    if (!genericStreams.empty()) {
      printer.printInfo() << "DEVICE::WARNING:" << genericStreams.size()
                               << "device generic stream(s) were not deleted.";
      for (auto stream : genericStreams) {
        hipStreamDestroy(stream); CHECK_ERR;
      }
    }
    status[StatusID::InterfaceInitialized] = false;
  }
}

int ConcreteAPI::getNumDevices() {
  int numDevices{};
  hipGetDeviceCount(&numDevices);
  CHECK_ERR;
  return numDevices;
}

int ConcreteAPI::getDeviceId() {
  if (!status[StatusID::DeviceSelected]) {
    logError() << "Device has not been selected. Please, select device before requesting device Id";
  }
  return currentDeviceId;
}

unsigned ConcreteAPI::getGlobMemAlignment() {
  // TODO: use hipDeviceGetAttribute
  return 128;
}

void ConcreteAPI::syncDevice() {
  isFlagSet<DeviceSelected>(status);
  hipDeviceSynchronize();
  CHECK_ERR;
}

std::string ConcreteAPI::getDeviceInfoAsText(int deviceId) {
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, deviceId);
  CHECK_ERR;

  std::ostringstream info;
  info << "Name: " << property.name << '\n';
  info << "totalGlobalMem: " << property.totalGlobalMem << '\n';
  info << "sharedMemPerBlock: " << property.sharedMemPerBlock << '\n';
  info << "regsPerBlock: " << property.regsPerBlock << '\n';
  info << "warpSize: " << property.warpSize << '\n';
  info << "memPitch: " << property.memPitch << '\n';
  info << "maxThreadsPerBlock: " << property.maxThreadsPerBlock << '\n';
  info << "totalConstMem: " << property.totalConstMem << '\n';
  info << "clockRate: " << property.clockRate << '\n';
  info << "multiProcessorCount: " << property.multiProcessorCount << '\n';
  info << "integrated: " << property.integrated << '\n';
  info << "canMapHostMemory: " << property.canMapHostMemory << '\n';
  info << "computeMode: " << property.computeMode << '\n';
  info << "concurrentKernels: " << property.concurrentKernels << '\n';
  info << "pciBusID: " << property.pciBusID << '\n';
  info << "pciDeviceID: " << property.pciDeviceID << '\n';

  return info.str();
}

std::string ConcreteAPI::getApiName() {
  return "CUDA";
}

std::string ConcreteAPI::getDeviceName(int deviceId) {
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, deviceId);
  CHECK_ERR;

  return property.name;
}

std::string ConcreteAPI::getPciAddress(int deviceId) {
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, deviceId);
  CHECK_ERR;

  std::ostringstream str;
  str << std::setfill('0') << std::setw(4) << std::hex << property.pciDomainID << ":" << std::setw(2) << property.pciBusID << ":" << property.pciDeviceID << "." << "0";
  return str.str();
}

void ConcreteAPI::putProfilingMark(const std::string &name, ProfilingColors color) {
#ifdef PROFILING_ENABLED
  isFlagSet<DeviceSelected>(status);
  nvtxEventAttributes_t eventAttrib = {0};
  eventAttrib.version = NVTX_VERSION;
  eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  eventAttrib.colorType = NVTX_COLOR_ARGB;
  eventAttrib.color = static_cast<uint32_t>(color);
  eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
  eventAttrib.message.ascii = name.c_str();
  nvtxRangePushEx(&eventAttrib);
#endif
}

void ConcreteAPI::popLastProfilingMark() {
#ifdef PROFILING_ENABLED
  isFlagSet<DeviceSelected>(status);
  nvtxRangePop();
#endif
}

void ConcreteAPI::setupPrinting(int rank) {
  printer.setRank(rank);
}

