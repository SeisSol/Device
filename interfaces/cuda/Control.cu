#include "utils/logger.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <string>

#ifdef PROFILING_ENABLED
#include <nvToolsExt.h>
#endif

#include "CudaWrappedAPI.h"
#include "Internals.h"

using namespace device;

ConcreteAPI::ConcreteAPI() {
  hipInit(0);
  CHECK_ERR;
  status[StatusId::DriverApiInitialized] = true;
}

void ConcreteAPI::setDevice(int deviceId) {
  currentDeviceId = deviceId;
  hipSetDevice(currentDeviceId);
  CHECK_ERR;

#ifndef DEVICE_USE_GRAPH_CAPTURING
  const auto id = currentDeviceId;
  logWarning(id) << "compute-graph capturing disabled for this device";
#endif

  status[StatusId::DeviceSelected] = true;
}

void ConcreteAPI::initialize() {
  if (!status[StatusId::DeviceSelected]) {
    logError() << "Device has not been selected. Please, select device before calling initialize";
  }
  if (!status[StatusId::InterfaceInitialized]) {

    hipStreamCreateWithFlags(&defaultStream, hipStreamNonBlocking); CHECK_ERR;
    hipEventCreate(&defaultStreamEvent); CHECK_ERR;

    constexpr size_t concurrencyLevel = 32;
    circularStreamBuffer.resize(concurrencyLevel);
    for (auto &stream : circularStreamBuffer) {
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking); CHECK_ERR;
      CHECK_ERR;
    }

    circularStreamEvents.resize(concurrencyLevel);
    for (auto &event : circularStreamEvents) {
      hipEventCreate(&event);
      CHECK_ERR;
    }

    status[StatusId::InterfaceInitialized] = true;
  }
  else {
    logWarning() << "Device Interface has already been initialized";
  }
}

void ConcreteAPI::allocateStackMem() {
  isFlagSet<StatusId::DeviceSelected>();

  // try to detect the amount of temp. memory from the environment
  const size_t factor = 1024 * 1024 * 1024; //!< bytes in 1 GB

  try {
    char *valueString = std::getenv("DEVICE_STACK_MEM_SIZE");
    const auto id = currentDeviceId;
    if (!valueString) {
      logInfo(id)
          << "From device: env. variable \"DEVICE_STACK_MEM_SIZE\" has not been set. "
          << "The default amount of the device memory (1 GB) "
          << "is going to be used to store temp. variables during execution of compute-algorithms.";
    } else {
      double requestedStackMem = std::stod(std::string(valueString));
      maxStackMem = factor * requestedStackMem;
      logInfo(id) << "From device: env. variable \"DEVICE_STACK_MEM_SIZE\" has been detected. "
                  << requestedStackMem << "GB of the device memory is going to be used "
                  << "to store temp. variables during execution of compute-algorithms.";
    }
  } catch (const std::invalid_argument &err) {
    logError() << "DEVICE::ERROR: " << err.what() << ". File: " << __FILE__
               << ", line: " << __LINE__;
  } catch (const std::out_of_range &err) {
    logError() << "DEVICE::ERROR: " << err.what() << ". File: " << __FILE__
               << ", line: " << __LINE__;
  }

  hipMalloc(&stackMemory, maxStackMem);
  CHECK_ERR;

  status[StatusId::StackMemAllocated] = true;
}


void ConcreteAPI::finalize() {
  if (status[StatusId::StackMemAllocated]) {
    hipFree(stackMemory);
    CHECK_ERR;
    stackMemory = nullptr;
    stackMemByteCounter = 0;
    stackMemMeter = std::stack<size_t>{};
    status[StatusId::StackMemAllocated] = false;

  }
  if (status[StatusId::InterfaceInitialized]) {
    // destroy default stream
    hipStreamDestroy(defaultStream);
    CHECK_ERR;

    // destroy default stream event
    hipEventDestroy(defaultStreamEvent);
    CHECK_ERR;

    // default circular streams
    for (auto &stream : circularStreamBuffer) {
      hipStreamDestroy(stream);
      CHECK_ERR;
    }
    circularStreamBuffer.clear();

    // destroy stream events
    for (auto &event : circularStreamEvents) {
      hipEventDestroy(event);
      CHECK_ERR;
    }
    circularStreamEvents.clear();

    // default captured graphs
    for (auto &graphInstance : graphs) {
      hipGraphExecDestroy(graphInstance.instance);
      CHECK_ERR;

      hipGraphDestroy(graphInstance.graph);
      CHECK_ERR;

      hipStreamDestroy(graphInstance.graphExecutionStream);
      CHECK_ERR;

      hipEventDestroy(graphInstance.graphCaptureEvent);
      CHECK_ERR;
    }
    graphs.clear();

    status[StatusId::InterfaceInitialized] = false;
  }
}


int ConcreteAPI::getNumDevices() {
  int numDevices{};
  hipGetDeviceCount(&numDevices);
  CHECK_ERR;
  return numDevices;
}

unsigned ConcreteAPI::getMaxThreadBlockSize() {
  int blockSize{};
  hipDeviceGetAttribute(&blockSize, hipDeviceAttributeMaxThreadsPerBlock, currentDeviceId);
  CHECK_ERR;
  return static_cast<unsigned>(blockSize);
}

unsigned ConcreteAPI::getMaxSharedMemSize() {
  int sharedMemSize{};
  hipDeviceGetAttribute(&sharedMemSize, hipDeviceAttributeMaxSharedMemoryPerBlock, currentDeviceId);
  CHECK_ERR;
  return static_cast<unsigned>(sharedMemSize);
}

unsigned ConcreteAPI::getGlobMemAlignment() {
  // TODO: use hipDeviceGetAttribute
  return 128;
}

void ConcreteAPI::synchDevice() {
  isFlagSet<DeviceSelected>();
  hipDeviceSynchronize();
  CHECK_ERR;
}

std::string ConcreteAPI::getDeviceInfoAsText(int deviceId) {
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, deviceId);
  CHECK_ERR;

  std::ostringstream info;
  info << "Name: " << property.name << '\n';
  info << "totalGlobalMem: " << property.totalGlobalMem << '\n';
  info << "sharedMemPerBlock: " << property.sharedMemPerBlock << '\n';
  info << "regsPerBlock: " << property.regsPerBlock << '\n';
  info << "warpSize: " << property.warpSize << '\n';
  info << "memPitch: " << property.memPitch << '\n';
  info << "maxThreadsPerBlock: " << property.maxThreadsPerBlock << '\n';
  info << "totalConstMem: " << property.totalConstMem << '\n';
  info << "clockRate: " << property.clockRate << '\n';
  info << "multiProcessorCount: " << property.multiProcessorCount << '\n';
  info << "integrated: " << property.integrated << '\n';
  info << "canMapHostMemory: " << property.canMapHostMemory << '\n';
  info << "computeMode: " << property.computeMode << '\n';
  info << "concurrentKernels: " << property.concurrentKernels << '\n';
  info << "pciBusID: " << property.pciBusID << '\n';
  info << "pciDeviceID: " << property.pciDeviceID << '\n';

  return info.str();
}

void ConcreteAPI::putProfilingMark(const std::string &name, ProfilingColors color) {
#ifdef PROFILING_ENABLED
  isFlagSet<DeviceSelected>();
  nvtxEventAttributes_t eventAttrib = {0};
  eventAttrib.version = NVTX_VERSION;
  eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  eventAttrib.colorType = NVTX_COLOR_ARGB;
  eventAttrib.color = static_cast<uint32_t>(color);
  eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
  eventAttrib.message.ascii = name.c_str();
  nvtxRangePushEx(&eventAttrib);
#endif
}

void ConcreteAPI::popLastProfilingMark() {
#ifdef PROFILING_ENABLED
  isFlagSet<DeviceSelected>();
  nvtxRangePop();
#endif
}
