#include "CudaWrappedAPI.h"
#include "Internals.h"
#include "utils/logger.h"
#include <algorithm>

using namespace device;

void ConcreteAPI::copyTo(void *dst, const void *src, size_t count) {
  hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
  CHECK_ERR;
  m_statistics.explicitlyTransferredDataToDeviceBytes += count;
}

void ConcreteAPI::copyFrom(void *dst, const void *src, size_t count) {
  hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
  CHECK_ERR;
  m_statistics.explicitlyTransferredDataToHostBytes += count;
}

void ConcreteAPI::copyBetween(void *dst, const void *src, size_t count) {
  hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice);
  CHECK_ERR;
}

void ConcreteAPI::copy2dArrayTo(void *dst, size_t dpitch, const void *src, size_t spitch,
                                size_t width, size_t height) {
  hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyHostToDevice);
  CHECK_ERR;
  m_statistics.explicitlyTransferredDataToDeviceBytes += width * height;
}

void ConcreteAPI::copy2dArrayFrom(void *dst, size_t dpitch, const void *src, size_t spitch,
                                  size_t width, size_t height) {
  hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToHost);
  CHECK_ERR;
  m_statistics.explicitlyTransferredDataToHostBytes += width * height;
}

void ConcreteAPI::prefetchUnifiedMemTo(Destination type, const void *devPtr, size_t count,
                                       void *streamPtr) {
  hipStream_t stream = (streamPtr == nullptr) ? 0 : (static_cast<hipStream_t>(streamPtr));
#ifndef NDEBUG
  auto itr = std::find(m_circularStreamBuffer.begin(), m_circularStreamBuffer.end(), stream);
  if (itr == m_circularStreamBuffer.end()) {
    logError() << "DEVICE::ERROR: passed stream does not belong to circular stream buffer";
  }
#endif
  hipMemPrefetchAsync(devPtr,
                       count,
                       type == Destination::CurrentDevice ? m_currentDeviceId : hipCpuDeviceId,
                       stream);
  CHECK_ERR;
}
