#include "hip/hip_runtime.h"
#include "utils/logger.h"
#include "CudaWrappedAPI.h"
#include "Internals.h"
#include <algorithm>

using namespace device;


void ConcreteAPI::copyTo(void* Dst, const void* Src, size_t Count) {
  hipMemcpy(Dst, Src, Count, hipMemcpyHostToDevice); CHECK_ERR;
  m_Statistics.ExplicitlyTransferredDataToDeviceBytes += Count;
}


void ConcreteAPI::copyFrom(void* Dst, const void* Src, size_t Count) {
  hipMemcpy(Dst, Src, Count, hipMemcpyDeviceToHost); CHECK_ERR;
  m_Statistics.ExplicitlyTransferredDataToHostBytes += Count;
}


void ConcreteAPI::copyBetween(void* Dst, const void* Src, size_t Count) {
  hipMemcpy(Dst, Src, Count, hipMemcpyDeviceToDevice); CHECK_ERR;
}


void ConcreteAPI::copy2dArrayTo(void *Dst,
                                size_t Dpitch,
                                const void *Src,
                                size_t Spitch,
                                size_t Width,
                                size_t Height) {
  hipMemcpy2D(Dst, Dpitch, Src, Spitch, Width, Height, hipMemcpyHostToDevice); CHECK_ERR;
  m_Statistics.ExplicitlyTransferredDataToDeviceBytes += Width * Height;
}


void ConcreteAPI::copy2dArrayFrom(void *Dst,
                                  size_t Dpitch,
                                  const void *Src,
                                  size_t Spitch,
                                  size_t Width,
                                  size_t Height) {
  hipMemcpy2D(Dst, Dpitch, Src, Spitch, Width, Height, hipMemcpyDeviceToHost); CHECK_ERR;
  m_Statistics.ExplicitlyTransferredDataToHostBytes += Width * Height;
}


__global__ void kernel_streamBatchedData(real **BaseSrcPtr,
                                         real **BaseDstPtr,
                                         unsigned ElementSize) {

  real *SrcElement = BaseSrcPtr[blockIdx.x];
  real *DstElement = BaseDstPtr[blockIdx.x];
  for (int Index = threadIdx.x; Index < ElementSize; Index += blockDim.x) {
    DstElement[Index] = SrcElement[Index];
  }
}

void ConcreteAPI::streamBatchedData(real **BaseSrcPtr,
                                    real **BaseDstPtr,
                                    unsigned ElementSize,
                                    unsigned NumElements) {
  dim3 Block(internals::WARP_SIZE, 1, 1);
  dim3 Grid(NumElements, 1, 1);
  kernel_streamBatchedData<<<Grid, Block>>>(BaseSrcPtr, BaseDstPtr, ElementSize); CHECK_ERR;
}

__global__ void kernel_accumulateBatchedData(real **BaseSrcPtr,
                                             real **BaseDstPtr,
                                             unsigned ElementSize) {

  real *SrcElement = BaseSrcPtr[blockIdx.x];
  real *DstElement = BaseDstPtr[blockIdx.x];
  for (int Index = threadIdx.x; Index < ElementSize; Index += blockDim.x) {
    DstElement[Index] += SrcElement[Index];
  }
}

void ConcreteAPI::accumulateBatchedData(real **BaseSrcPtr,
                                        real **BaseDstPtr,
                                        unsigned ElementSize,
                                        unsigned NumElements) {
  dim3 Block(internals::WARP_SIZE, 1, 1);
  dim3 Grid(NumElements, 1, 1);
  kernel_accumulateBatchedData<<<Grid, Block>>>(BaseSrcPtr, BaseDstPtr, ElementSize); CHECK_ERR;
}


void ConcreteAPI::prefetchUnifiedMemTo(Destination Type, const void* DevPtr, size_t Count, void* streamPtr) {
  hipStream_t stream = (streamPtr == nullptr) ? 0 : *(static_cast<hipStream_t*>(streamPtr));
#ifndef NDEBUG
  auto itr = std::find(m_circularStreamBuffer.begin(), m_circularStreamBuffer.end(), stream);
  if (itr == m_circularStreamBuffer.end()) {
    logError() << "DEVICE::ERROR: passed stream does not belong to circular stream buffer";
  }
#endif
  hipMemPrefetchAsync(DevPtr,
                       Count,
                       Type == Destination::CurrentDevice ? m_CurrentDeviceId : hipCpuDeviceId,
                       stream); CHECK_ERR;
}
