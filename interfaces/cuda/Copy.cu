// SPDX-FileCopyrightText: 2020-2024 SeisSol Group
//
// SPDX-License-Identifier: BSD-3-Clause

#include "CudaWrappedAPI.h"
#include "Internals.h"
#include "utils/logger.h"
#include <algorithm>
#include <cassert>

using namespace device;

void ConcreteAPI::copyTo(void *dst, const void *src, size_t count) {
  isFlagSet<DeviceSelected>(status);
  hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
  CHECK_ERR;
  statistics.explicitlyTransferredDataToDeviceBytes += count;
}

void ConcreteAPI::copyFrom(void *dst, const void *src, size_t count) {
  isFlagSet<DeviceSelected>(status);
  hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
  CHECK_ERR;
  statistics.explicitlyTransferredDataToHostBytes += count;
}

void ConcreteAPI::copyBetween(void *dst, const void *src, size_t count) {
  isFlagSet<DeviceSelected>(status);
  hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice);
  CHECK_ERR;
}

void ConcreteAPI::copyToAsync(void *dst, const void *src, size_t count, void* streamPtr) {
  isFlagSet<InterfaceInitialized>(status);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  hipMemcpyAsync(dst, src, count, hipMemcpyHostToDevice, stream);
  CHECK_ERR;
}

void ConcreteAPI::copyFromAsync(void *dst, const void *src, size_t count, void* streamPtr) {
  isFlagSet<InterfaceInitialized>(status);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToHost, stream);
  CHECK_ERR;
}

void ConcreteAPI::copyBetweenAsync(void *dst, const void *src, size_t count, void* streamPtr) {
  isFlagSet<InterfaceInitialized>(status);
  hipStream_t stream = (streamPtr != nullptr) ? static_cast<hipStream_t>(streamPtr) : 0;
  hipMemcpyAsync(dst, src, count, hipMemcpyDeviceToDevice, stream);
  CHECK_ERR;
}

void ConcreteAPI::copy2dArrayTo(void *dst, size_t dpitch, const void *src, size_t spitch,
                                size_t width, size_t height) {
  isFlagSet<DeviceSelected>(status);
  hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyHostToDevice);
  CHECK_ERR;
  statistics.explicitlyTransferredDataToDeviceBytes += width * height;
}

void ConcreteAPI::copy2dArrayFrom(void *dst, size_t dpitch, const void *src, size_t spitch,
                                  size_t width, size_t height) {
  isFlagSet<DeviceSelected>(status);
  hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToHost);
  CHECK_ERR;
  statistics.explicitlyTransferredDataToHostBytes += width * height;
}

void ConcreteAPI::prefetchUnifiedMemTo(Destination type, const void *devPtr, size_t count,
                                       void *streamPtr) {
  isFlagSet<InterfaceInitialized>(status);
  hipStream_t stream = (streamPtr == nullptr) ? 0 : (static_cast<hipStream_t>(streamPtr));
  hipMemPrefetchAsync(devPtr,
                       count,
                       type == Destination::CurrentDevice ? currentDeviceId : hipCpuDeviceId,
                       stream);
  CHECK_ERR;
}

