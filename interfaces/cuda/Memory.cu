#include <assert.h>
#include <iostream>
#include <sstream>
#include <cstring>
#include <stdexcept>

#include "CudaWrappedAPI.h"
#include "Internals.h"

using namespace device;

void *ConcreteAPI::allocGlobMem(size_t size) {
  isFlagSet<DeviceSelected>(status);
  void *devPtr;
  hipMalloc(&devPtr, size);
  CHECK_ERR;
  statistics.allocatedMemBytes += size;
  memToSizeMap[devPtr] = size;
  return devPtr;
}

namespace device {
hipError_t setProp(hipMemAllocationProp *prop) {
  hipDevice_t currentDevice;
  if (hipCtxGetDevice(&currentDevice) != hipSuccess)
    return hipErrorOutOfMemory;

  std::memset(prop, 0, sizeof(hipMemAllocationProp));
  prop->type = hipMemAllocationTypePinned;
  prop->location.type = hipMemLocationTypeDevice;
  prop->location.id = currentDevice;
  prop->allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  return hipSuccess;
}
}

void *ConcreteAPI::allocCompressibleGlobMem(size_t size) {
  void* addr{ nullptr};
  if (useCompressibleMemory) {
    isFlagSet<DeviceSelected>(status);
    try {
      std::stringstream errStream;

      hipMemAllocationProp prop = {};
      hipError_t err = setProp(&prop);

      if (err != hipSuccess) {
        errStream << "could not set device properties";
        throw std::runtime_error(errStream.str());
      }

      size_t granularity = 0;
      hipError_t result = hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
      if (result != hipSuccess) {
        errStream << "could not calculates granularity. Error: " << result;
        throw std::runtime_error(std::to_string(result));
      }
      size = ((size - 1) / granularity + 1) * granularity;

      hipDeviceptr_t dptr;
      result = hipMemAddressReserve(&dptr, size, 0, 0, 0);
      if (result != hipSuccess) {
        errStream << "could not reserve memory address. Error: " << result;
        throw std::runtime_error(errStream.str());
      }

      hipMemGenericAllocationHandle_t allocationHandle;
      result = hipMemCreate(&allocationHandle, size, &prop, 0);
      if (result != hipSuccess) {
        errStream << "could not create compressible memory. Error: " << result;
        throw std::runtime_error(errStream.str());
      }

      hipMemAllocationProp allocationProp = {};
      hipMemGetAllocationPropertiesFromHandle(&allocationProp, allocationHandle);
      if (allocationProp.allocFlags.compressionType != CU_MEM_ALLOCATION_COMP_GENERIC) {
        errStream << "hipMemCreate was not able to allocate compressible memory";
        throw std::runtime_error(errStream.str());
      }

      result = hipMemMap(dptr, size, 0, allocationHandle, 0);
      if (result != hipSuccess) {
        errStream << "could not map compressible memory. Error: " << result;
        throw std::runtime_error(errStream.str());
      }

      result = hipMemRelease(allocationHandle);
      if (result != hipSuccess) {
        errStream << "could not release a memory handle representing a memory allocation. Error: " << result;
        throw std::runtime_error(errStream.str());
      }

      hipMemAccessDesc accessDescriptor;
      accessDescriptor.location.id = prop.location.id;
      accessDescriptor.location.type = prop.location.type;
      accessDescriptor.flags = hipMemAccessFlagsProtReadWrite;

      result = hipMemSetAccess(dptr, size, &accessDescriptor, 1);
      if (result != hipSuccess) {
        errStream << "could not set the access flags. Error: " << result;
        throw std::runtime_error(errStream.str());
      }

      addr = reinterpret_cast<void*>(dptr);
      compressibleMemSizesTable[addr] = size;
    }
    catch (std::runtime_error& err) {
      std::stringstream stream;
      stream << "failed to allocate compressible memory because: " << err.what();
      throw std::runtime_error(stream.str());
    }
  }
  else {
    addr = this->allocGlobMem(size);
  }
  return addr;
}

void *ConcreteAPI::allocUnifiedMem(size_t size) {
  isFlagSet<DeviceSelected>(status);
  void *devPtr;
  hipMallocManaged(&devPtr, size, hipMemAttachGlobal);
  CHECK_ERR;
  if (allowedConcurrentManagedAccess) {
    hipMemAdvise(devPtr, size, hipMemAdviseSetPreferredLocation, currentDeviceId);
    CHECK_ERR;
  }
  statistics.allocatedMemBytes += size;
  statistics.allocatedUnifiedMemBytes += size;
  memToSizeMap[devPtr] = size;
  return devPtr;
}

void *ConcreteAPI::allocPinnedMem(size_t size) {
  isFlagSet<DeviceSelected>(status);
  void *devPtr;
  hipHostMalloc(&devPtr, size);
  CHECK_ERR;
  statistics.allocatedMemBytes += size;
  memToSizeMap[devPtr] = size;
  return devPtr;
}

void ConcreteAPI::freeMem(void *devPtr) {
  isFlagSet<DeviceSelected>(status);
  assert((memToSizeMap.find(devPtr) != memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  statistics.deallocatedMemBytes += memToSizeMap[devPtr];
  hipFree(devPtr);
  CHECK_ERR;
}

void ConcreteAPI::freeCompressibleMem(void *devPtr) {
  if (useCompressibleMemory) {
    isFlagSet<DeviceSelected>(status);
    try {
      if (devPtr != nullptr) {
        std::stringstream errStream;

        hipMemAllocationProp prop = {};
        hipError_t err = setProp(&prop);

        if (err != hipSuccess) {
          errStream << "could not set device properties";
          throw std::runtime_error(errStream.str());
        }

        auto size = compressibleMemSizesTable.at(devPtr);
        hipError_t result = hipMemUnmap((hipDeviceptr_t)devPtr, size);
        if (result != hipSuccess) {
          errStream << "could not unmap the backing memory of a given address range. Error: " << result;
          throw std::runtime_error(errStream.str());
        }

        result = hipMemAddressFree((hipDeviceptr_t)devPtr, size);
        if (result != hipSuccess) {
          errStream << "could not free an address range reservation. Error: " << result;
          throw std::runtime_error(errStream.str());
        }
      }
    }
    catch (std::runtime_error& err) {
      std::stringstream stream;
      stream << "failed to deallocate compressible memory because: " << err.what();
      throw std::runtime_error(stream.str());
    }
  }
  else {
    return this->freeMem(devPtr);
  }
}

void ConcreteAPI::freePinnedMem(void *devPtr) {
  isFlagSet<DeviceSelected>(status);
  assert((memToSizeMap.find(devPtr) != memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  statistics.deallocatedMemBytes += memToSizeMap[devPtr];
  hipHostFree(devPtr);
  CHECK_ERR;
}

char *ConcreteAPI::getStackMemory(size_t requestedBytes) {
  isFlagSet<StackMemAllocated>(status);
  assert(((stackMemByteCounter + requestedBytes) < maxStackMem) &&
         "DEVICE:: run out of a device stack memory");
  char *mem = &stackMemory[stackMemByteCounter];

  size_t requestedAlignedBytes = align(requestedBytes, getGlobMemAlignment());
  stackMemByteCounter += requestedAlignedBytes;
  stackMemMeter.push(requestedAlignedBytes);
  return mem;
}

void ConcreteAPI::popStackMemory() {
  isFlagSet<StackMemAllocated>(status);
  stackMemByteCounter -= stackMemMeter.top();
  stackMemMeter.pop();
}

std::string ConcreteAPI::getMemLeaksReport() {
  isFlagSet<DeviceSelected>(status);
  std::ostringstream report{};
  report << "Memory Leaks, bytes: "
         << (statistics.allocatedMemBytes - statistics.deallocatedMemBytes) << '\n';
  report << "Stack Memory Leaks, bytes: " << stackMemByteCounter << '\n';
  return report.str();
}

size_t ConcreteAPI::getMaxAvailableMem() {
  isFlagSet<DeviceSelected>(status);
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, currentDeviceId);
  CHECK_ERR;
  return property.totalGlobalMem;
}

size_t ConcreteAPI::getCurrentlyOccupiedMem() {
  isFlagSet<DeviceSelected>(status);
  return statistics.allocatedMemBytes;
}

size_t ConcreteAPI::getCurrentlyOccupiedUnifiedMem() {
  isFlagSet<DeviceSelected>(status);
  return statistics.allocatedUnifiedMemBytes;
}
