// SPDX-FileCopyrightText: 2020-2024 SeisSol Group
//
// SPDX-License-Identifier: BSD-3-Clause

#include <assert.h>
#include <cstring>
#include <hip/driver_types.h>
#include <iostream>
#include <sstream>

#include "AbstractAPI.h"
#include "CudaWrappedAPI.h"
#include "Internals.h"

#include "utils/logger.h"

#include <hip/hip_runtime.h>

using namespace device;

namespace {

// adapted from https://github.com/NVIDIA/cuda-samples/blob/master/Samples/3_CUDA_Features/cudaCompressibleMemory/compMalloc.cpp

void* driverAllocate(std::size_t size, const hipMemAllocationProp& prop) {
  std::size_t granularity = 1;
  hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
  CHECK_ERR;

  size = (size + granularity - 1) / size;

  hipDeviceptr_t cptr;
  hipMemAddressReserve(&cptr, size, 0, 0, 0);
  CHECK_ERR;
  // cf. https://docs.nvidia.com/cuda/hopper-tuning-guide/index.html?highlight=inline%20compression#inline-compression
  hipMemGenericAllocationHandle_t allocationHandle;
  hipMemCreate(&allocationHandle, size, &prop, 0);
  CHECK_ERR;
  hipMemMap(cptr, size, 0, allocationHandle, 0);
  CHECK_ERR;
  hipMemRelease(allocationHandle);
  CHECK_ERR;

  return reinterpret_cast<void*>(cptr);
}

void driverFree(void* ptr, std::size_t size, const hipMemAllocationProp& prop) {
  std::size_t granularity = 1;
  hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
  CHECK_ERR;

  size = (size + granularity - 1) / size;

  hipDeviceptr_t cptr = reinterpret_cast<hipDeviceptr_t>(ptr);

  hipMemUnmap(cptr, size);
  hipMemAddressFree(cptr, size);
}

} // namespace

void *ConcreteAPI::allocGlobMem(size_t size, bool compress) {
  isFlagSet<DeviceSelected>(status);
  void *devPtr;
  if (compress && canCompress) {
    hipMemAllocationProp prop = {};
    std::memset(&prop, 0, sizeof(hipMemAllocationProp));
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = currentDeviceId;
    prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

    devPtr = driverAllocate(size, prop);
    allocationProperties[devPtr] = reinterpret_cast<void*>(new hipMemAllocationProp(prop));
  }
  else {
    hipMalloc(&devPtr, size);
    CHECK_ERR;
  }
  statistics.allocatedMemBytes += size;
  memToSizeMap[devPtr] = size;
  return devPtr;
}

void *ConcreteAPI::allocUnifiedMem(size_t size, bool compress, Destination hint) {
  isFlagSet<DeviceSelected>(status);
  void *devPtr;
  hipMallocManaged(&devPtr, size, hipMemAttachGlobal);
  CHECK_ERR;
  if (hint == Destination::Host) {
    hipMemAdvise(devPtr, size, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    CHECK_ERR;
  }
  else if (allowedConcurrentManagedAccess) {
    hipMemAdvise(devPtr, size, hipMemAdviseSetPreferredLocation, currentDeviceId);
    CHECK_ERR;
  }
  statistics.allocatedMemBytes += size;
  statistics.allocatedUnifiedMemBytes += size;
  memToSizeMap[devPtr] = size;
  return devPtr;
}

void *ConcreteAPI::allocPinnedMem(size_t size, bool compress, Destination hint) {
  isFlagSet<DeviceSelected>(status);
  void *devPtr;
  const auto flag = hint == Destination::Host ? hipHostMallocDefault : hipHostMallocMapped;
  hipHostAlloc(&devPtr, size, flag);
  CHECK_ERR;
  statistics.allocatedMemBytes += size;
  memToSizeMap[devPtr] = size;
  return devPtr;
}

void ConcreteAPI::freeGlobMem(void *devPtr) {
  isFlagSet<DeviceSelected>(status);
  assert((memToSizeMap.find(devPtr) != memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  statistics.deallocatedMemBytes += memToSizeMap[devPtr];
  if (allocationProperties.find(devPtr) != allocationProperties.end()) {
    driverFree(devPtr, memToSizeMap.at(devPtr), *reinterpret_cast<hipMemAllocationProp*>(allocationProperties.at(devPtr)));
  }
  else {
    hipFree(devPtr);
    CHECK_ERR;
  }
}

void ConcreteAPI::freeUnifiedMem(void *devPtr) {
  isFlagSet<DeviceSelected>(status);
  assert((memToSizeMap.find(devPtr) != memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  statistics.deallocatedMemBytes += memToSizeMap[devPtr];
  hipFree(devPtr);
  CHECK_ERR;
}

void ConcreteAPI::freePinnedMem(void *devPtr) {
  isFlagSet<DeviceSelected>(status);
  assert((memToSizeMap.find(devPtr) != memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  statistics.deallocatedMemBytes += memToSizeMap[devPtr];
  hipHostFree(devPtr);
  CHECK_ERR;
}

void *ConcreteAPI::allocMemAsync(size_t size, void* streamPtr) {
  if (size == 0) {
    return nullptr;
  }
  else {
    void* ptr;
    hipMallocAsync(&ptr, size, static_cast<hipStream_t>(streamPtr));
    CHECK_ERR;
    return ptr;
  }
}
void ConcreteAPI::freeMemAsync(void *devPtr, void* streamPtr) {
  if (devPtr != nullptr) {
    hipFreeAsync(ptr, static_cast<hipStream_t>(streamPtr));
    CHECK_ERR;
  }
}

std::string ConcreteAPI::getMemLeaksReport() {
  isFlagSet<DeviceSelected>(status);
  std::ostringstream report{};
  report << "Memory Leaks, bytes: "
         << (statistics.allocatedMemBytes - statistics.deallocatedMemBytes) << '\n';
  return report.str();
}

size_t ConcreteAPI::getMaxAvailableMem() {
  isFlagSet<DeviceSelected>(status);
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, currentDeviceId);
  CHECK_ERR;
  return property.totalGlobalMem;
}

size_t ConcreteAPI::getCurrentlyOccupiedMem() {
  isFlagSet<DeviceSelected>(status);
  return statistics.allocatedMemBytes;
}

size_t ConcreteAPI::getCurrentlyOccupiedUnifiedMem() {
  isFlagSet<DeviceSelected>(status);
  return statistics.allocatedUnifiedMemBytes;
}

void ConcreteAPI::pinMemory(void* ptr, size_t size) {
  isFlagSet<DeviceSelected>(status);
  hipHostRegister(ptr, size, 0);
  CHECK_ERR;
}

void ConcreteAPI::unpinMemory(void* ptr) {
  isFlagSet<DeviceSelected>(status);
  hipHostUnregister(ptr);
  CHECK_ERR;
}

void* ConcreteAPI::devicePointer(void* ptr) {
  isFlagSet<DeviceSelected>(status);
  void* result;
  hipHostGetDevicePointer(&result, ptr, 0);
  CHECK_ERR;
  return result;
}

