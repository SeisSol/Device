#include <assert.h>
#include <iostream>
#include <sstream>

#include "CudaWrappedAPI.h"
#include "Internals.h"

using namespace device;

void *ConcreteAPI::allocGlobMem(size_t size) {
  void *devPtr;
  hipMalloc(&devPtr, size);
  CHECK_ERR;
  m_statistics.allocatedMemBytes += size;
  m_memToSizeMap[devPtr] = size;
  return devPtr;
}

void *ConcreteAPI::allocUnifiedMem(size_t size) {
  void *devPtr;
  hipMallocManaged(&devPtr, size, hipMemAttachGlobal);
  CHECK_ERR;
  m_statistics.allocatedMemBytes += size;
  m_statistics.allocatedUnifiedMemBytes += size;
  m_memToSizeMap[devPtr] = size;
  return devPtr;
}

void *ConcreteAPI::allocPinnedMem(size_t size) {
  void *devPtr;
  hipHostMalloc(&devPtr, size);
  CHECK_ERR;
  m_statistics.allocatedMemBytes += size;
  m_memToSizeMap[devPtr] = size;
  return devPtr;
}

void ConcreteAPI::freeMem(void *devPtr) {
  assert((m_memToSizeMap.find(devPtr) != m_memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  m_statistics.deallocatedMemBytes += m_memToSizeMap[devPtr];
  hipFree(devPtr);
  CHECK_ERR;
}

void ConcreteAPI::freePinnedMem(void *devPtr) {
  assert((m_memToSizeMap.find(devPtr) != m_memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  m_statistics.deallocatedMemBytes += m_memToSizeMap[devPtr];
  hipHostFree(devPtr);
  CHECK_ERR;
}

char *ConcreteAPI::getStackMemory(size_t requestedBytes) {
  assert(((m_stackMemByteCounter + requestedBytes) < m_maxStackMem) &&
         "DEVICE:: run out of a device stack memory");
  char *mem = &m_stackMemory[m_stackMemByteCounter];
  m_stackMemByteCounter += requestedBytes;
  m_stackMemMeter.push(requestedBytes);
  return mem;
}

void ConcreteAPI::popStackMemory() {
  m_stackMemByteCounter -= m_stackMemMeter.top();
  m_stackMemMeter.pop();
}

std::string ConcreteAPI::getMemLeaksReport() {
  std::ostringstream report{};
  report << "Memory Leaks, bytes: "
         << (m_statistics.allocatedMemBytes - m_statistics.deallocatedMemBytes) << '\n';
  report << "Stack Memory Leaks, bytes: " << m_stackMemByteCounter << '\n';
  return report.str();
}

size_t ConcreteAPI::getMaxAvailableMem() {
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, m_currentDeviceId);
  CHECK_ERR;
  return property.totalGlobalMem;
}

size_t ConcreteAPI::getCurrentlyOccupiedMem() { return m_statistics.allocatedMemBytes; }

size_t ConcreteAPI::getCurrentlyOccupiedUnifiedMem() {
  return m_statistics.allocatedUnifiedMemBytes;
}