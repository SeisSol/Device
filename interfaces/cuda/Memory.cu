// SPDX-FileCopyrightText: 2020-2024 SeisSol Group
//
// SPDX-License-Identifier: BSD-3-Clause

#include <assert.h>
#include <hip/driver_types.h>
#include <iostream>
#include <sstream>

#include "AbstractAPI.h"
#include "CudaWrappedAPI.h"
#include "Internals.h"

#include "utils/logger.h"

using namespace device;

void *ConcreteAPI::allocGlobMem(size_t size, bool compress) {
  isFlagSet<DeviceSelected>(status);
  void *devPtr;
  hipMalloc(&devPtr, size);
  CHECK_ERR;
  statistics.allocatedMemBytes += size;
  memToSizeMap[devPtr] = size;
  return devPtr;
}

void *ConcreteAPI::allocUnifiedMem(size_t size, bool compress, Destination hint) {
  isFlagSet<DeviceSelected>(status);
  void *devPtr;
  hipMallocManaged(&devPtr, size, hipMemAttachGlobal);
  CHECK_ERR;
  if (hint == Destination::Host) {
    hipMemAdvise(devPtr, size, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    CHECK_ERR;
  }
  else if (allowedConcurrentManagedAccess) {
    hipMemAdvise(devPtr, size, hipMemAdviseSetPreferredLocation, currentDeviceId);
    CHECK_ERR;
  }
  statistics.allocatedMemBytes += size;
  statistics.allocatedUnifiedMemBytes += size;
  memToSizeMap[devPtr] = size;
  return devPtr;
}

void *ConcreteAPI::allocPinnedMem(size_t size, bool compress, Destination hint) {
  isFlagSet<DeviceSelected>(status);
  void *devPtr;
  const auto flag = hint == Destination::Host ? hipHostMallocDefault : hipHostMallocMapped;
  hipHostAlloc(&devPtr, size, flag);
  CHECK_ERR;
  statistics.allocatedMemBytes += size;
  memToSizeMap[devPtr] = size;
  return devPtr;
}

void ConcreteAPI::freeGlobMem(void *devPtr) {
  isFlagSet<DeviceSelected>(status);
  assert((memToSizeMap.find(devPtr) != memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  statistics.deallocatedMemBytes += memToSizeMap[devPtr];
  hipFree(devPtr);
  CHECK_ERR;
}

void ConcreteAPI::freeUnifiedMem(void *devPtr) {
  isFlagSet<DeviceSelected>(status);
  assert((memToSizeMap.find(devPtr) != memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  statistics.deallocatedMemBytes += memToSizeMap[devPtr];
  hipFree(devPtr);
  CHECK_ERR;
}

void ConcreteAPI::freePinnedMem(void *devPtr) {
  isFlagSet<DeviceSelected>(status);
  assert((memToSizeMap.find(devPtr) != memToSizeMap.end()) &&
         "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  statistics.deallocatedMemBytes += memToSizeMap[devPtr];
  hipHostFree(devPtr);
  CHECK_ERR;
}

char *ConcreteAPI::getStackMemory(size_t requestedBytes) {
  isFlagSet<StackMemAllocated>(status);
  char *mem = &stackMemory[stackMemByteCounter];

  size_t requestedAlignedBytes = align(requestedBytes, getGlobMemAlignment());

  if ((stackMemByteCounter + requestedAlignedBytes) >= maxStackMem) {
    logError() << "DEVICE:: run out of device stack memory";
  }

  stackMemByteCounter += requestedAlignedBytes;
  stackMemMeter.push(requestedAlignedBytes);
  return mem;
}

void ConcreteAPI::popStackMemory() {
  isFlagSet<StackMemAllocated>(status);
  stackMemByteCounter -= stackMemMeter.top();
  stackMemMeter.pop();
}

std::string ConcreteAPI::getMemLeaksReport() {
  isFlagSet<DeviceSelected>(status);
  std::ostringstream report{};
  report << "Memory Leaks, bytes: "
         << (statistics.allocatedMemBytes - statistics.deallocatedMemBytes) << '\n';
  report << "Stack Memory Leaks, bytes: " << stackMemByteCounter << '\n';
  return report.str();
}

size_t ConcreteAPI::getMaxAvailableMem() {
  isFlagSet<DeviceSelected>(status);
  hipDeviceProp_t property;
  hipGetDeviceProperties(&property, currentDeviceId);
  CHECK_ERR;
  return property.totalGlobalMem;
}

size_t ConcreteAPI::getCurrentlyOccupiedMem() {
  isFlagSet<DeviceSelected>(status);
  return statistics.allocatedMemBytes;
}

size_t ConcreteAPI::getCurrentlyOccupiedUnifiedMem() {
  isFlagSet<DeviceSelected>(status);
  return statistics.allocatedUnifiedMemBytes;
}

void ConcreteAPI::pinMemory(void* ptr, size_t size) {
  isFlagSet<DeviceSelected>(status);
  hipHostRegister(ptr, size, 0);
  CHECK_ERR;
}

void ConcreteAPI::unpinMemory(void* ptr) {
  isFlagSet<DeviceSelected>(status);
  hipHostUnregister(ptr);
  CHECK_ERR;
}

void* ConcreteAPI::devicePointer(void* ptr) {
  isFlagSet<DeviceSelected>(status);
  void* result;
  hipHostGetDevicePointer(&result, ptr, 0);
  CHECK_ERR;
  return result;
}

