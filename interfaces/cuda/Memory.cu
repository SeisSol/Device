#include "hip/hip_runtime.h"
#include <assert.h>
#include <sstream>
#include <iostream>

#include "CudaWrappedAPI.h"
#include "Internals.h"

using namespace device;

void* ConcreteAPI::allocGlobMem(size_t Size) {
  void *DevPtr;
  hipMalloc(&DevPtr, Size); CHECK_ERR;
  m_Statistics.AllocatedMemBytes += Size;
  m_MemToSizeMap[DevPtr] = Size;
  return DevPtr;
}


void* ConcreteAPI::allocUnifiedMem(size_t Size) {
  void *DevPtr;
  hipMallocManaged(&DevPtr, Size, hipMemAttachGlobal); CHECK_ERR;
  m_Statistics.AllocatedMemBytes += Size;
  m_Statistics.AllocatedUnifiedMemBytes += Size;
  m_MemToSizeMap[DevPtr] = Size;
  return DevPtr;
}


void* ConcreteAPI::allocPinnedMem(size_t Size) {
  void *DevPtr;
  hipHostMalloc(&DevPtr, Size); CHECK_ERR;
  m_Statistics.AllocatedMemBytes += Size;
  m_MemToSizeMap[DevPtr] = Size;
  return DevPtr;
}


void ConcreteAPI::freeMem(void *DevPtr) {
  assert((m_MemToSizeMap.find(DevPtr) != m_MemToSizeMap.end())
          && "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  m_Statistics.DeallocatedMemBytes += m_MemToSizeMap[DevPtr];
  hipFree(DevPtr); CHECK_ERR;
}


void ConcreteAPI::freePinnedMem(void *DevPtr) {
  assert((m_MemToSizeMap.find(DevPtr) != m_MemToSizeMap.end())
         && "DEVICE: an attempt to delete mem. which has not been allocated. unknown pointer");
  m_Statistics.DeallocatedMemBytes += m_MemToSizeMap[DevPtr];
  hipHostFree(DevPtr); CHECK_ERR;
}


char* ConcreteAPI::getStackMemory(size_t RequestedBytes) {
  assert(((m_StackMemByteCounter + RequestedBytes) < m_MaxStackMem) && "DEVICE:: run out of a device stack memory");
  char *Mem = &m_StackMemory[m_StackMemByteCounter];
  m_StackMemByteCounter += RequestedBytes;
  m_StackMemMeter.push(RequestedBytes);
  return Mem;
}


void ConcreteAPI::popStackMemory() {
  m_StackMemByteCounter -= m_StackMemMeter.top();
  m_StackMemMeter.pop();
}

std::string ConcreteAPI::getMemLeaksReport() {
  std::ostringstream Report{};
  Report << "Memory Leaks, bytes: " << (m_Statistics.AllocatedMemBytes - m_Statistics.DeallocatedMemBytes) << '\n';
  Report << "Stack Memory Leaks, bytes: " << m_StackMemByteCounter << '\n';
  return Report.str();
}


__global__ void kernel_touchMemory(real *Ptr, size_t Size, bool Clean) {
  int Id = threadIdx.x + blockIdx.x * blockDim.x;
  if (Id < Size) {
    if (Clean) {
      Ptr[Id] = 0.0;
    }
    else {
      real Value = Ptr[Id];
      // Do something dummy here. We just need to check the pointers point to valid memory locations.
      // Avoid compiler optimization. Possibly, implement a dummy code with asm.
      Value += 1.0;
    }
  }
}

void ConcreteAPI::touchMemory(real *Ptr, size_t Size, bool Clean) {
  dim3 Block(256, 1, 1);
  dim3 Grid = internals::computeGrid1D(Block, Size);
  kernel_touchMemory<<<Grid, Block>>>(Ptr, Size, Clean); CHECK_ERR;
}


__global__ void kernel_touchBatchedMemory(real **BasePtr, unsigned ElementSize, bool Clean) {
  real *Element = BasePtr[blockIdx.x];
  int Id = threadIdx.x;
  while (Id < ElementSize) {
    if (Clean) {
      Element[Id] = 0.0;
    } else {
      real Value = Element[Id];
      // Do something dummy here. We just need to check the pointers point to valid memory locations.
      // Avoid compiler optimization. Possibly, implement a dummy code with asm.
      Value += 1.0;
    }
    Id += blockDim.x;
  }
}

void ConcreteAPI::touchBatchedMemory(real **BasePtr, unsigned ElementSize, unsigned NumElements, bool Clean) {
  dim3 Block(256, 1, 1);
  dim3 Grid(NumElements, 1, 1);
  kernel_touchBatchedMemory<<<Grid, Block>>>(BasePtr, ElementSize, Clean); CHECK_ERR;
}

size_t ConcreteAPI::getMaxAvailableMem() {
  hipDeviceProp_t Property;
  hipGetDeviceProperties(&Property, m_CurrentDeviceId); CHECK_ERR;
  return Property.totalGlobalMem;
}

size_t ConcreteAPI::getCurrentlyOccupiedMem() {
  return m_Statistics.AllocatedMemBytes;
}

size_t ConcreteAPI::getCurrentlyOccupiedUnifiedMem() {
  return m_Statistics.AllocatedUnifiedMemBytes;
}